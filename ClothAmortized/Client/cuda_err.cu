#include "hip/hip_runtime.h"
#include "cuda_err.hh"

#include <stdio.h>

void exit_on_err(hipError_t err) {
	char * err_str;
	switch (err) {
		case hipSuccess                      : return;
		case hipErrorMissingConfiguration    : err_str = "Missing configuration error."      ; break;
		case hipErrorOutOfMemory        : err_str = "Memory allocation error."          ; break;
		case hipErrorNotInitialized     : err_str = "Initialization error."             ; break;
		case hipErrorLaunchFailure           : err_str = "Launch failure."                   ; break;
		case hipErrorPriorLaunchFailure      : err_str = "Prior launch failure."             ; break;
		case hipErrorLaunchTimeOut           : err_str = "Launch timeout error."             ; break;
		case hipErrorLaunchOutOfResources    : err_str = "Launch out of resources error."    ; break;
		case hipErrorInvalidDeviceFunction   : err_str = "Invalid device function."          ; break;
		case hipErrorInvalidConfiguration    : err_str = "Invalid configuration."            ; break;
		case hipErrorInvalidDevice           : err_str = "Invalid device."                   ; break;
		case hipErrorInvalidValue            : err_str = "Invalid value."                    ; break;
		case hipErrorInvalidPitchValue       : err_str = "Invalid pitch value."              ; break;
		case hipErrorInvalidSymbol           : err_str = "Invalid symbol."                   ; break;
		case hipErrorMapFailed   : err_str = "Map buffer object failed."         ; break;
		case hipErrorUnmapFailed : err_str = "Unmap buffer object failed."       ; break;
		case cudaErrorInvalidHostPointer      : err_str = "Invalid host pointer."             ; break;
		case hipErrorInvalidDevicePointer    : err_str = "Invalid device pointer."           ; break;
		case hipErrorInvalidTexture          : err_str = "Invalid texture."                  ; break;
		case cudaErrorInvalidTextureBinding   : err_str = "Invalid texture binding."          ; break;
		case hipErrorInvalidChannelDescriptor: err_str = "Invalid channel descriptor."       ; break;
		case hipErrorInvalidMemcpyDirection  : err_str = "Invalid memcpy direction."         ; break;
		case cudaErrorAddressOfConstant       : err_str = "Address of constant error."        ; break;
		case cudaErrorTextureFetchFailed      : err_str = "Texture fetch failed."             ; break;
		case cudaErrorTextureNotBound         : err_str = "Texture not bound error."          ; break;
		case cudaErrorSynchronizationError    : err_str = "Synchronization error."            ; break;
		case cudaErrorInvalidFilterSetting    : err_str = "Invalid filter setting."           ; break;
		case cudaErrorInvalidNormSetting      : err_str = "Invalid norm setting."             ; break;
		case cudaErrorMixedDeviceExecution    : err_str = "Mixed device execution."           ; break;
		case hipErrorDeinitialized         : err_str = "CUDA runtime unloading."           ; break;
		case hipErrorUnknown                 : err_str = "Unknown error condition."          ; break;
		case cudaErrorNotYetImplemented       : err_str = "Function not yet implemented."     ; break;
		case cudaErrorMemoryValueTooLarge     : err_str = "Memory value too large."           ; break;
		case hipErrorInvalidHandle   : err_str = "Invalid resource handle."          ; break;
		case hipErrorNotReady                : err_str = "Not ready error."                  ; break;
		case hipErrorInsufficientDriver      : err_str = "CUDA runtime is newer than driver."; break;
		case hipErrorSetOnActiveProcess      : err_str = "Set on active process error."      ; break;
		case hipErrorNoDevice                : err_str = "No available CUDA device."         ; break;
		case cudaErrorStartupFailure          : err_str = "Startup failure."                  ; break;
		case cudaErrorApiFailureBase          : err_str = "API failure base."                 ; break;
	}
	printf("Error: %s\n", err_str);
	exit(-1);
}
