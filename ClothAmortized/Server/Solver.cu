#include "hip/hip_runtime.h"
// 
//  Solver.cpp
//  Cloth Simulation
//  
//  Created by Timothy Luciani on 2011-03-12.
//  Copyright 2011 __MyCompanyName__. All rights reserved.
// 

#include "Particle.hh"

#include "open_gl.hh"
#include "cuda_err.hh"
#include "cuda_helpers.hh"
#include <hip/hip_runtime.h>

#include <cutil.h>
#include <cuda_gl_interop.h>

#include <vector>
#include <cstdlib>
#include <cstdio>

#define TIME_STEP 0.5*0.5 // how large time step each particle takes each frame
#define CONSTRAINT_ITERATIONS 25 // how many iterations of constraint satisfaction each frame (more is rigid, less is soft)

const float3 gravity = make_float3(0.0f, -0.15f, 0.0f);
static const int threadsPerBlock = 256;

__device__ __host__
int getParticle(int x, int y, int row){ return y*row+x; }

/* find the normal of a triangle */
__device__
float3  triangle_normal(int v1, int v2, int v3, float *points)
{
	float3 p1 = make_float3(points[v1*3 + 0], points[v1*3 + 1], points[v1*3 + 2]);
	float3 p2 = make_float3(points[v2*3 + 0], points[v2*3 + 1], points[v2*3 + 2]);
	float3 p3 = make_float3(points[v3*3 + 0], points[v3*3 + 1], points[v3*3 + 2]);
	
	return (cross(p2-p1, p3-p1));
}

__device__
float3 triangle_normal(float3 v1, float3 v2, float3 v3){ return ( cross(v2-v1, v3-v1) ); }

/* apply the wind force to the cloth */

__device__
float3 windForce(struct Particle *pVector, float3 windDir, int x, int y, int row)
{
	float3 normal = triangle_normal(pVector[getParticle(x+1,y,row)].m_Position, 
		pVector[getParticle(x,y,row)].m_Position, pVector[getParticle(x,y+1,row)].m_Position);
	
	float3 d = normalize(normal);
	float3 force = normal * dot(d,windDir);
	
	normal = triangle_normal(pVector[getParticle(x+1,y+1,row)].m_Position, 
	pVector[getParticle(x+1,y,row)].m_Position, pVector[getParticle(x,y+1,row)].m_Position);

	d = normalize(normal);
	force += normal * dot(d,windDir);

	return force;
}

__global__
void add_force(struct Particle *pVector, float3 gravity, bool wind, int row, int column){	
	
	// calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int x = index%row;
	int y = index/column;
	
	/* gravity */
	pVector[index].addForce(gravity * TIME_STEP);
	
	if(wind && y < (column -1) && x < (row-1)){
		float3 windDir = make_float3(0.3f, 0.3f, 0.2f);
	
		/* wind */
		pVector[index].addForce( windForce(pVector, windDir, x, y, row) * 20 );
	
	}
	
	pVector[index].step(TIME_STEP);
	
}

__global__
void satisfy(struct Particle *pVector, float *data_pointer, int row, int column){
	
	//calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	 
	int ii = index%row;
	int jj = index/column;

	for(int i = 0; i < CONSTRAINT_ITERATIONS; i++){		
		
		if(ii < row-1){ // to the right

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii+1, jj, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}

		}
		if(jj < column -1){ // below	

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii, jj+1, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}

		}
		if(ii < row-1 && jj < column -1){ // down right

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii+1, jj+1, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}	
		}
		if(ii < row-1 && jj < column -1){ // up right	

			int p1 = getParticle(ii+1,jj, row);
			int p2 = getParticle(ii, jj+1, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}	
		}
		/* neighbor's neighbors */

	 	if(ii < row-2){ // to the right

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii+2, jj, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos; 
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}	
		}
		if(jj < column -2){ // below	

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii, jj+2, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}
		}

		if(ii < row-2 && jj < column -2){ // down right

			int p1 = getParticle(ii,jj, row);
			int p2 = getParticle(ii+2, jj+2, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}
		}

		if(ii < row-2 && jj < column -2){ // up right	

			int p1 = getParticle(ii+2,jj, row);
			int p2 = getParticle(ii, jj+2, row);

			float3 diff = pVector[p1].m_ConstructPos - pVector[p2].m_ConstructPos;
			float m_rest = length(diff);

			float3 p1_to_p2 = pVector[p2].m_Position - pVector[p1].m_Position; 
			float current_distance = length(p1_to_p2);
			float3 correctionVector = p1_to_p2*(1 - m_rest/current_distance); 
			float3 correctionVectorHalf = correctionVector*0.5; 

			if(current_distance > m_rest){

				pVector[p1].updateVector(correctionVectorHalf, data_pointer); 
				pVector[p2].updateVector(-correctionVectorHalf, data_pointer);

			} // end if

			if(!pVector[p1].m_movable){
				
				pVector[p1].m_Position = pVector[p1].m_ConstructPos;
				data_pointer[p1 * 3 + 0] = pVector[p1].m_ConstructPos.x;
				data_pointer[p1 * 3 + 1] = pVector[p1].m_ConstructPos.y;
				data_pointer[p1 * 3 + 2] = pVector[p1].m_ConstructPos.z;
				
			}
			if(!pVector[p2].m_movable){
				
				pVector[p2].m_Position = pVector[p2].m_ConstructPos;
				data_pointer[p2 * 3 + 0] = pVector[p2].m_ConstructPos.x;
				data_pointer[p2 * 3 + 1] = pVector[p2].m_ConstructPos.y;
				data_pointer[p2 * 3 + 2] = pVector[p2].m_ConstructPos.z;
			}
		}	
	}
}

__global__
void calculate_flag_normals(float *data_pointer, float *flagNorms, int row, int column)
{
	// calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int x = index%row;
	int y = index/column;
	
	// normal of current particle
    float3 currNorm = make_float3(0.0f, 0.0f, 0.0f);

    if(x == 0 && y == 0)
    {
        // Top Left
        currNorm += triangle_normal(index, index+column, index+1, data_pointer);
    }
    else if(x == (column-1) && y == 0)
    {
        // Top Right
        currNorm += triangle_normal(index, index-1, index+column-1, data_pointer);
        currNorm += triangle_normal(index, index+column-1, index+column, data_pointer);
    }
    else if(x == 0 && y == (column-1))
    {
        // Bottom Left
        currNorm += triangle_normal(index, index-column+1, index-column, data_pointer);
        currNorm += triangle_normal(index, index+1, index-column+1, data_pointer);
    }
    else if(x == (row-1) && y == (column-1))
    {
        // Bottom Right
        currNorm += triangle_normal(index, index-column, index-1, data_pointer);
    }
    else if(y == 0)
    {
        // Top row
        currNorm += triangle_normal(index, index-1, index+column-1, data_pointer);
        currNorm += triangle_normal(index, index+column-1, index+column, data_pointer);
        currNorm += triangle_normal(index, index+column, index+1, data_pointer);
    }
    else if(y == (column-1))
    {
        // Bottom row
        currNorm += triangle_normal(index, index-column, index-1, data_pointer);
        currNorm += triangle_normal(index, index-column+1, index-column, data_pointer);
        currNorm += triangle_normal(index, index+1, index-column+1, data_pointer);
    }
    else if(x == 0)
    {
        // Left column
        currNorm += triangle_normal(index, index-column+1, index-column, data_pointer);
        currNorm += triangle_normal(index, index+1, index-column+1, data_pointer);
        currNorm += triangle_normal(index, index+column, index+1, data_pointer);
    }
    else if(x == (row-1))
    {
        // Right column 
        currNorm += triangle_normal(index, index-column, index-1, data_pointer);
        currNorm += triangle_normal(index, index-1, index+column-1, data_pointer);
        currNorm += triangle_normal(index, index+column-1, index+column, data_pointer);
    }
    else
    {
        // Middle vertex that touches six faces
        currNorm += triangle_normal(index, index-1, index+column-1, data_pointer);
        currNorm += triangle_normal(index, index+column-1, index+column, data_pointer);
        currNorm += triangle_normal(index, index+column, index+1, data_pointer);
		currNorm += triangle_normal(index, index-column, index-1, data_pointer);
        currNorm += triangle_normal(index, index-column+1, index-column, data_pointer);
        currNorm += triangle_normal(index, index+1, index-column+1, data_pointer);
    }

    // Normalize the normal of this vertex and write it to normal pointer
	currNorm = normalize(currNorm);
    flagNorms[index*3 + 0] = currNorm.x;
	flagNorms[index*3 + 1] = currNorm.y;
	flagNorms[index*3 + 2] = currNorm.z;
}

void verlet_simulation_step(struct Particle* pVector, float *data_pointer, float *norms, bool wind, int row, int column){
				
	/* set up number of threads to run */	
	int totalThreads = row * column;
	int nBlocks = totalThreads/threadsPerBlock;
	nBlocks += ((totalThreads % threadsPerBlock) > 0) ? 1 : 0;
	
	/* apply wind and gravity forces */	
	add_force<<<nBlocks, threadsPerBlock>>>(pVector, gravity, wind, row, column);
		
	hipDeviceSynchronize();
					
    // Satisfy constraints
 	satisfy<<<nBlocks, threadsPerBlock>>>(pVector, data_pointer, row, column);

	hipDeviceSynchronize();
	
    // Calculate flag normals
	calculate_flag_normals<<<nBlocks, threadsPerBlock>>>(data_pointer, norms, row, column);
	
	hipDeviceSynchronize();
				
} // end sim step
