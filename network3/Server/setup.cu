#include "hip/hip_runtime.h"
// 
//  setup.cu
//  Cloth Simulation
//  
//  Created by Timothy Luciani on 2011-04-10.
//  Modified by Mitch Luban on 2011-04-12
//  Copyright 2011 __MyCompanyName__. All rights reserved.
// 

#include "imageio.hh"
#include "Particle.hh"

#include "setup.hh"

#include <cutil.h>
#include <cuda_gl_interop.h>

#include <stdlib.h>
#include <stdio.h>
#include <cmath>

#define BLACK   0
#define RED     1
#define YELLOW  2
#define MAGENTA 3
#define GREEN   4
#define CYAN    5
#define BLUE    6
#define GREY    7
#define WHITE   8

#define MAXSAMPLES 100

extern int numCloths;

const int threadsPerBlock = 256;
extern void verlet_simulation_step(struct Particle* pVector, float *data_pointer, bool wind, int row, int column);

int row	  = 40;
int column = 40;
unsigned int numTriangles = (row-1)*(column-1)*2;

int width = 8;
int height = 4;

struct Particle* pVector;

//GLuint texVbo;
//GLuint indexVbo;
unsigned int *flagIndexArray;
float *data_pointer;
float *gpuData_pointer;

float2 *flagTexArray;
GLuint flagTexId;

int size = row * column;

extern bool dsim;
extern bool wind;

__device__
int getParticleInd(int x, int y, int row){
	return x + y * row;
}

float *get_dataPtr(){
	return data_pointer;
}

uint *get_indexPtr()
{
	return flagIndexArray;
}

/*----------------------------------------------------------------------
free/clear/allocate simulation data
----------------------------------------------------------------------*/
void free_data ( void )
{
	hipFree(pVector);
	
	free(data_pointer);
//    glDeleteBuffers(1, &indexVbo);
//    glDeleteBuffers(1, &texVbo);

    free(flagIndexArray);
    free(flagTexArray);
}

/*--------------------------------------------------------------------
					Make Particles
--------------------------------------------------------------------*/
__global__
void make_particles(struct Particle *pVector, float *data_pointer, int row, int column, int width, int height)
{
	// //calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// 
	int i = index%row;
	int j = index/column;
	
	float3 pos = make_float3(width * (i/(float)row), -height * (j/(float)column), 0);
	
	if((j == 0 && i == 0) || (i == 0 && j == column-1))
		pVector[getParticleInd(i,j,row)] = Particle(pos, 1, data_pointer, getParticleInd(i,j, row), false);
	else
		pVector[getParticleInd(i,j,row)] = Particle(pos, 1, data_pointer, getParticleInd(i,j, row), true);
	
} // end make particles

/*--------------------------------------------------------------------
					Make Flag Mesh out of particles
--------------------------------------------------------------------*/

void make_flag_mesh( void )
{
    unsigned int currIndex = 0;

    float colFloat = (float)(column-1);
    float rowFloat = (float)(row-1);

    for(unsigned int ii = 0; ii < (size - column); ii++)
    {
        if( (ii+1) % column == 0 )
            continue;

        flagIndexArray[currIndex + 0] = ii + 0;
        flagIndexArray[currIndex + 1] = ii + column;
        flagIndexArray[currIndex + 2] = ii + 1;

        currIndex += 3;
    }

    for(unsigned int ii = row; ii < size; ii++)
    {
        if( (ii+1) % column == 0 )
            continue;

        flagIndexArray[currIndex + 0] = ii + 0;
        flagIndexArray[currIndex + 1] = ii + 1;
        flagIndexArray[currIndex + 2] = (ii + 1) - column;

        currIndex += 3;
    }

    for(unsigned int ii = 0; ii < size; ii++)
    {
        int currX = column - ii%column;
        int currY = (ii/column)%row;
        flagTexArray[ii] = make_float2((float)currX/colFloat, (float)(currY)/rowFloat);
    }
}

/*--------------------------------------------------------------------
					Initialize System
--------------------------------------------------------------------*/

void init_system(void)
{		
    data_pointer = (float*)malloc(sizeof(float) * size * 3);
	hipMalloc((void**)&gpuData_pointer, sizeof(float) * size * 3);
	
	hipMalloc( (void**)&(pVector), size * sizeof(struct Particle) );
	   
    /* create and copy */
    int totalThreads = row * column;
    int nBlocks = totalThreads/threadsPerBlock;
    nBlocks += ((totalThreads % threadsPerBlock) > 0) ? 1 : 0;

    make_particles<<<nBlocks, threadsPerBlock>>>(pVector, gpuData_pointer, row, column, width, height); // create particles

	hipMemcpy(data_pointer, gpuData_pointer, sizeof(float) * size * 3, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
	
    /******************************
     * Flag texturing and meshing
     * ***************************/

    flagIndexArray = (unsigned int*)malloc(sizeof(unsigned int) * numTriangles * 3);
    flagTexArray = (float2*)malloc(sizeof(float2) * size);
    make_flag_mesh();

    // glGenBuffers(1, &indexVbo);
    //    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, indexVbo);
    //    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(unsigned int) * numTriangles * 3, flagIndexArray, GL_STATIC_DRAW);
    // 
    //    glGenBuffers(1, &texVbo);
    //    glBindBuffer(GL_ARRAY_BUFFER, texVbo);
    //    glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * size, flagTexArray, GL_STATIC_DRAW);

    const char *flagTextureFilename = "Textures/american_flag.png";
    int w, h;
    unsigned char *data = loadImageRGBA(flagTextureFilename, &w, &h);

    // glGenTextures(1, &flagTexId);
    //     glActiveTexture(GL_TEXTURE0_ARB);
    //     glBindTexture(GL_TEXTURE_2D, flagTexId);
    // 
    //     glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    //     glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    //     glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    //     glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // 
    //     glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE, data);
    
    free(data);
}
/*----------------------------------------------------------------------
relates mouse movements to tinker toy construction
----------------------------------------------------------------------*/
__global__
void remap_GUI(struct Particle *pVector, float *data_pointer)
{	
	// //calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
		
	pVector[index].reset();
	data_pointer[index * 3 + 0] = pVector[index].m_ConstructPos.x;
	data_pointer[index * 3 + 1] = pVector[index].m_ConstructPos.y;
	data_pointer[index * 3 + 2] = pVector[index].m_ConstructPos.z;
	
}

void step_func ( )
{
 	if ( dsim ){ // simulate
	    verlet_simulation_step(pVector, gpuData_pointer, wind, row, column);
		hipMemcpy(data_pointer, gpuData_pointer, sizeof(float) * size * 3, hipMemcpyDeviceToHost);
	}
    else { // remap

	    int totalThreads = row * column;
	    int nBlocks = totalThreads/threadsPerBlock;
	    nBlocks += ((totalThreads % threadsPerBlock) > 0) ? 1 : 0;
	
	    remap_GUI<<<nBlocks, threadsPerBlock>>>(pVector, gpuData_pointer);
		hipMemcpy(data_pointer, gpuData_pointer, sizeof(float) * size * 3, hipMemcpyDeviceToHost);
		
         hipDeviceSynchronize();
    }
    
}
